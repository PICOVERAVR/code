
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void matmul(float* Md, float* Nd, float* outd, int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	float outval = 0.0;
	for (int n = 0; n < width; n++) {
		float Melem = Md[y * width + n];
		float Nelem = Nd[x + width * n];
		outval += Melem * Nelem;
	}
	outd[y * width + x] = outval;
}

void printDeviceInfo() {
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    int driver;
    hipDriverGetVersion(&driver);
    
    int runtime;
    hipRuntimeGetVersion(&runtime);
    
    cout << "GPU: " << prop.name << endl;
    cout << "Driver Version: " << driver << endl;
    cout << "Runtime Version: " << runtime << endl << endl;
}

int main(int argc, char **argv) {
	printDeviceInfo();
	
	const float A[] = {
		1.0, 0.0, 0.0, 0.0,
		0.0, 1.0, 0.0, 0.0,
		0.0, 0.0, 1.0, 0.0,
		0.0, 0.0, 0.0, 1.0,
	};

	const float B[] = {
		1.0, 0.0, 0.0, 0.0,
		0.0, 1.0, 0.0, 0.0,
		0.0, 0.0, 1.0, 0.0,
		0.0, 0.0, 0.0, 1.0,
	};
	
	float C[16] = { 0.0 };
	
	float* Ad;
	float* Bd;
	float* Cd;
	
	const unsigned int width = 4;	
	const unsigned int size = width * width * sizeof(float);
	
	hipMalloc((void**) &Ad, size);
	hipMalloc((void**) &Bd, size);
	hipMalloc((void**) &Cd, size);

	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
	
	dim3 dimBlock(width, width, 1);
	dim3 dimGrid(1, 1, 1); // dim3 = vec3, 1 for final element because grids have to be 2D
	
	matmul<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, width);
	
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
	
	for (size_t i = 0; i < 16; i++) {
		if ((i % 4 == 0) && (i > 0)) {
			cout << endl;
		}
		cout << C[i] << " ";
	}
	cout << endl;
	
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
}
